#include "hip/hip_runtime.h"
#include "shared.h"
#include <hipblas.h>
#include <stdio.h>
#include <cutil_inline.h>
#define EPSILON 0.000001

extern "C" int genome_index;

MATRIX<MATRIX_TYPE> device_ww2, device_save, device_sum, device_scratch;
MATRIX<unsigned int> device_labels, device_indices,device_ww_count, device_ret,device_ww_count2;
ORDERED_MATRIX<MATRIX_TYPE, COLUMN_MAJOR> device_ww, device_data;


unsigned int *ret, *indices;

float host_alpha[2];
int host_r = -1, host_beta[2];

__constant__ uint constant_color[COLOR_SIZE];


__global__ void calc_ww2(const MATRIX<MATRIX_TYPE> ww, MATRIX_TYPE *ww2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int j=0; j<VECTOR_SIZE; j++){
		//this shouldn't be backwards...*sigh*
		ww2[i] += pow(ww.data[i * ww.row + j ],2);
	}
}

__global__ void update_weights(float *a, float *b, uint *ww_count, uint *count, int _beta)
{
	int row = threadIdx.x + blockDim.x * blockIdx.x;
	int col = threadIdx.y + blockDim.y * blockIdx.y;
	int index = row + IMAGE_M * col;
	if (col < IMAGE_M){
		int imin = max(row - _beta, 0);
		int imax = min(row + _beta + 1, IMAGE_N);

		for (int x=imin; x<imax; x++){
			for (int k=0; k<VECTOR_SIZE; k++){
				b[k + VECTOR_SIZE * ( row + IMAGE_M * col )] += a[k + VECTOR_SIZE * (x + IMAGE_M * col)];
			}
			count[index] += ww_count[x + IMAGE_M * col];
		}

//		for (int i=0; i<VECTOR_SIZE; i++){  //vector size...
//			for (int k= _min; k<_max; k++){
//				b[i * IMAGE_MxN + index]  += a[i * IMAGE_MxN + k * IMAGE_M + col];
//			}
//		}
//		for (int k= _min; k<_max; k++){
//			count[index] += ww_count[k * IMAGE_M + col];
//		}
	}
}

__global__ void update_weights2(float *ww, float *a, float *b, uint *ww_count, uint *count, int _beta, float _alpha)
{
	int row = threadIdx.x + blockDim.x * blockIdx.x;
	int col = threadIdx.y + blockDim.y * blockIdx.y;
	int index = row * IMAGE_M + col;
//	__shared__ float s_ww[IMAGE_N * IMAGE_M];
	int _min = max(col - _beta, 0);
	int _max = min(col + _beta + 1, IMAGE_M);

	if (col < IMAGE_M){
		int imin = max(row - _beta,0);
		int imax = min(row + _beta + 1,IMAGE_N);
		float sum = 0;
		for (int x=imin; x<imax; x++){
			for (int k=0; k<VECTOR_SIZE; k++){
				a[k + VECTOR_SIZE * ( col + IMAGE_M * row) ] += b[k + VECTOR_SIZE * ( col + IMAGE_M * x) ];
			}
			ww_count[col + IMAGE_M * row] += count[col + IMAGE_M * x];
		}
	}
//		for (int i=0; i<VECTOR_SIZE; i++){  //vector size...
//			for (int k= _min; k<_max; k++){
//				a[i * IMAGE_MxN + index]  += b[i * IMAGE_MxN + row * IMAGE_M + k];
//			}
//		}
//		for (int k= _min; k<_max; k++){
//			ww_count[index] += count[row * IMAGE_M + k];
//		}
//
//		for (int i=0; i<VECTOR_SIZE; i++){
//			if (ww_count[index] == 0)
//				a[i * IMAGE_MxN + index] = 0;
//			else
//				a[ i * IMAGE_MxN + index]  /= (float)ww_count[index];
//        	ww[i * IMAGE_MxN + index] = abs(ww[i * IMAGE_MxN + index]  +_alpha * (a[i * IMAGE_MxN + index] - ww[i * IMAGE_MxN + index]));
//		}
//		for (int i=0; i<VECTOR_SIZE; i++){
//	    	ww[index] += ww[i * IMAGE_MxN + index];
//		}
//		for (int i=0; i<VECTOR_SIZE; i++){
//			if (ww[index] > 0)
//				ww[i * IMAGE_MxN + index] /= (float)ww[index];
//			else
//				ww[i * IMAGE_MxN + index] = 0;
//
//		}
//	}
//	__syncthreads();
//	if (col < IMAGE_M){
//		for (int i=0; i<VECTOR_SIZE; i++){
//			if (ww[index] > 0)
//				ww[i * IMAGE_MxN + index] /= (float)ww[index];
//			else
//				ww[i * IMAGE_MxN + index] = 0;
//
//		}
//	}
}

//Calculate argmax and sum the data vectors
__global__ void reduce(uint *ret, uint *indices, float *ww_sum, const float *vec, const float *data, int index)
{
	int size = 1024;
	//using shared memory here will limit me...
	//initialize with hard coded numbers because compile error on variable initialization
	__shared__ int argmax[1024];
	__shared__ float s_vec[1024];

	int blocksize = REDUCE_BLOCKSIZE;
	int coalesce_num = size/blocksize;

	for (int i=0; i<1024/REDUCE_BLOCKSIZE; i++){
		argmax[threadIdx.x + i * blocksize] = threadIdx.x + i * blocksize;
		s_vec[threadIdx.x + i * blocksize] = vec[threadIdx.x + i * blocksize];
	}


	// Large number ->32
	for (int j=1; j < coalesce_num; j++){
		if (threadIdx.x + blocksize * j < IMAGE_MxN){
			argmax[threadIdx.x] = (s_vec[argmax[threadIdx.x]] > s_vec[argmax[j * blocksize + threadIdx.x]])?
						argmax[threadIdx.x]:argmax[j * blocksize + threadIdx.x];
		}
	}

	//32->16, 16->8, 8->4, 4->2, 2->1
	for (int i=0; i<LOG2_REDUCE_BLOCKSIZE; i++){
		__syncthreads();
		blocksize = blocksize/2;

		if (threadIdx.x < blocksize){
			argmax[threadIdx.x] = s_vec[ argmax[blocksize +threadIdx.x]] < s_vec[argmax[threadIdx.x]]? argmax[threadIdx.x]:(argmax[blocksize+threadIdx.x]);
		}
	}
	__syncthreads();
	if (threadIdx.x < 1){
		ret[ argmax[0] ]++;
		indices[index] = argmax[0];
	}
	//take the vector from data and save it to ww_sum
	if (threadIdx.x < VECTOR_SIZE)
		ww_sum[ argmax[0] *VECTOR_SIZE + threadIdx.x] += data[index * VECTOR_SIZE + threadIdx.x];//ww_sum[ 410 * 16 + threadIdx.x] = data[threadIdx.x];
}

__global__ void buildImage(uint *im, uint *labels, uint *indices)
{
	uint i = threadIdx.x + blockDim.x * blockIdx.x;
	__shared__ int nn[IMAGE_N];
	__shared__ int mm[IMAGE_N];
	nn[threadIdx.x] = indices[i] / IMAGE_M;
	mm[threadIdx.x] = indices[i] - IMAGE_M * nn[threadIdx.x];
	im[ nn[threadIdx.x] * IMAGE_M + mm[threadIdx.x]] = labels[i];
}

__global__ void buildSplitImage(uint *im, uint *labels, uint *indices, int g_index)
{
	uint tidx = threadIdx.x + blockDim.x * blockIdx.x;
	uint tidy = threadIdx.y + blockDim.y * blockIdx.y;
	uint index = tidx * IMAGE_N + tidy;

	int genome[GENOMIC_DATA_COUNT];

	for (int i=0; i<GENOMIC_DATA_COUNT; i++)
		genome[i] = 0;

	for (int i=0; i<DATA_SIZE; i++){
		if (indices[i] == index){
			genome[ labels[i] ]++;
		}
	}

	int count = 0;
	for (int i=0; i<GENOMIC_DATA_COUNT; i++){
		count = 0;
		for (int j=0; j<GENOMIC_DATA_COUNT; j++){
			if (i != j)
				count += (genome[i] > genome[j]);
		}
		if (count == (GENOMIC_DATA_COUNT - 1)){
			im[index] = genome[g_index];
			return;
		}
	}
	im[index] = 0;
//	if (genome[0] > genome[1] && genome[0] > genome[2] && genome[0] > genome[3])
//		im[index] = genome[g_index];
//	else if (genome[1] > genome[0] && genome[1] > genome[2] && genome[1] > genome[3])
//		im[index] = genome[g_index];
//	else if (genome[2] > genome[0] && genome[2] > genome[1] && genome[2] > genome[3])
//		im[index] = genome[g_index];
//	else if (genome[3] > genome[0] && genome[3] > genome[1] && genome[3] > genome[2])
//		im[index] = genome[g_index];
//	else
//		im[index] = 0;
}

__global__ void expandSplitImage(uint *im, const uint *ret)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	for (int i=0; i<16; i++){
		for (int j=0; j<16; j++){
			im[(y * 16 + j) * 512 + x * 16 + i] = LUMINANCE_ADJUSTMENT * ret[y * IMAGE_M + x];
		}
	}
}

__global__ void expandLogImage(unsigned char *im, const uint *ret)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	for (int i=0; i<16; i++){
		for (int j=0; j<16; j++){
			im[(y * 16 + j) * 512 + x * 16 + i] = 10 * logf(ret[y * IMAGE_M + x]);
		}
	}
}
__global__ void expandConstantImage(uint *im, const uint *ret)
{

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	for (int i=0; i<16; i++){
		for (int j=0; j<16; j++){
			im[(y * 16 + j) * 512 + x * 16 + i] = constant_color[ret[y * IMAGE_M + x]] * ret[y * IMAGE_M + x];
		}
	}
}

extern "C" void generateSplitImage(int g_index, unsigned int * device_split_pbo)
{
	dim3 block(16,16);
	dim3 grid(IMAGE_M/16,IMAGE_N/16);
	expandSplitImage<<<grid,block>>>(device_split_pbo, device_ret.data + g_index * IMAGE_MxN);
}

extern "C" void cleanup()
{
    hipFree (device_ww.data);
    hipFree (device_data.data);
    hipFree (device_ww2.data);
	hipFree(device_indices.data);
	hipFree(device_labels.data);
	delete  ret, indices;
}
extern "C" void setupCuda(ORDERED_MATRIX<MATRIX_TYPE, COLUMN_MAJOR> ww,  ORDERED_MATRIX<MATRIX_TYPE, ROW_MAJOR> data, uint *labels, unsigned int *device_regular_pbo, uint *device_split_pbo, unsigned char *device_log_pbo)
{
    //setup color
	unsigned char color[COLOR_SIZE * 4];
	for(unsigned int i=0; i<COLOR_SIZE * 4; i+=4){
		color[i + 1] = (unsigned char)i;
		color[i + 2] = (i + 64) % 256;
		color[i + 3] = (i + 128) % 256;
		color[i] = (i + 192) % 256;
	}

	color[0] = 0;
	color[1] = 0;
	color[2] = 0;
	color[3] = 0;

	color[4] = 255;
	color[5] = 0;
	color[6] = 0;
	color[7] = 0;

	color[8] = 0;
	color[9] = 255;
	color[10] = 0;
	color[11] = 0;

	color[12] = 0;
	color[13] = 0;
	color[14] = 255;
	color[15] = 0;

	color[16] = 255;
	color[17] = 255;
	color[18] = 0;
	color[19] = 0;

	color[20] = 255;
	color[21] = 0;
	color[22] = 255;
	color[23] = 0;

	color[24] = 0;
	color[25] = 255;
	color[26] = 255;
	color[27] = 0;

	color[28] = 128;
	color[29] = 128;
	color[30] = 128;
	color[31] = 0;

	color[32] = 255;
	color[33] = 255;
	color[34] = 255;
	color[35] = 0;

	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(constant_color), color, sizeof(unsigned int) * COLOR_SIZE, 0));

	host_beta[0] = 8;
	host_beta[1] = 8;
	host_alpha[0] = .6;
	host_alpha[1] = .6;


	hipMemset(device_regular_pbo, 128, sizeof(unsigned int) * 512 * 512);
	hipMemset(device_split_pbo, 128, sizeof(unsigned int) * 512 * 512);
	hipMemset(device_log_pbo, 128, sizeof(unsigned char) * 512 * 512);

	device_labels.row = data.row;
	device_labels.col = 1;
	cutilSafeCall(hipMalloc((void**)&device_labels.data, sizeof(uint) * data.row));
	cutilSafeCall(hipMemcpy(device_labels.data, labels, sizeof(uint) * data.row, hipMemcpyHostToDevice));

	device_ww_count.row = 1024;
	device_ww_count.col = 1;
	cutilSafeCall(hipMalloc((void**)&device_ww_count.data, sizeof(unsigned int) * device_ww_count.row));
    cutilSafeCall(hipMemset((void*)device_ww_count.data, 0, sizeof(unsigned int) * device_ww_count.row));

	device_ww_count2.row = 1024;
	device_ww_count2.col = 1;
	cutilSafeCall(hipMalloc((void**)&device_ww_count2.data, sizeof(unsigned int) * device_ww_count2.row));
    cutilSafeCall(hipMemset((void*)device_ww_count2.data, 0, sizeof(unsigned int) * device_ww_count2.row));

    //multiply by the number of genomes
    //+1 for the regular image
    device_ret.row = ww.row * (GENOMIC_DATA_COUNT + 1);
    device_ret.col = 1;
	cutilSafeCall(hipMalloc((void**)&device_ret.data, sizeof(unsigned int) * device_ret.row));
    cutilSafeCall(hipMemset((void*)device_ret.data, 0, sizeof(unsigned int) * device_ret.row));

    device_indices.row = DATA_SIZE;
    device_indices.col = 1;
    cutilSafeCall(hipMalloc((void**)&device_indices.data, sizeof(unsigned int) * DATA_SIZE));
    cutilSafeCall(hipMemset((void*)device_indices.data, 0, sizeof(unsigned int) * DATA_SIZE));

    device_ww.row = ww.row;
    device_ww.col = ww.col;
    printf("setup matrix ww %d %d\n", ww.row, ww.col);
    cutilSafeCall(hipMalloc((void**)&device_ww.data, sizeof(float) * ww.row * ww.col));
    cutilSafeCall(hipMemcpy(device_ww.data, ww.data, sizeof(float) * ww.row * ww.col, hipMemcpyHostToDevice));

	device_ww2.row = IMAGE_N;
	device_ww2.col = IMAGE_M;
    printf("setup matrix ww2 %d %d\n", device_ww2.row, device_ww2.col);
    cutilSafeCall(hipMalloc((void**)&device_ww2.data, sizeof(float) * device_ww2.row * device_ww2.col));
	cutilSafeCall(hipMemset(device_ww2.data, 0, sizeof(float) * device_ww2.row * device_ww2.col));

    device_sum.row = ww.row;
    device_sum.col = ww.col;
    printf("setup matrix sum %d %d\n", device_sum.row, device_sum.col);
    cutilSafeCall(hipMalloc((void**)&device_sum.data, sizeof(float) * device_sum.row * device_sum.col));
    cutilSafeCall(hipMemset(device_sum.data, 0, sizeof(float) * device_sum.row * device_sum.col ));


    printf("setup matrix scractch %d %d\n", IMAGE_MxN, VECTOR_SIZE);
    cutilSafeCall(hipMalloc((void**)&device_scratch.data, sizeof(float) * IMAGE_MxN * VECTOR_SIZE));
    cutilSafeCall(hipMemset(device_scratch.data, 0, sizeof(float) * IMAGE_MxN * VECTOR_SIZE));

    device_save.row = device_ww2.row;
    device_save.col = device_ww2.col;
    device_save.data = device_scratch.data;


	ret = (unsigned int*)malloc(sizeof(unsigned int) * ww.row);
	indices = (uint*)malloc(sizeof(uint) * data.row);

    device_data.row = data.row;
    device_data.col = data.col;
    printf("setup matrix data %d %d\n", device_data.row, device_data.col);
    cutilSafeCall(hipMalloc((void**)&device_data.data, sizeof(float) * device_data.row*device_data.col));
    cutilSafeCall(hipMemcpy(device_data.data, data.data, sizeof(float) * device_data.row * device_data.col, hipMemcpyHostToDevice));
}

extern "C" int runCuda(unsigned int *device_regular_pbo, unsigned int *device_split_pbo, unsigned char *device_log_pbo)
{
	unsigned int timer;
    cutCreateTimer(&timer);
    double time,total_time;

    dim3 block;
    dim3 grid;

    total_time = 0;
    cutResetTimer(timer);
    cutStartTimer(timer);
    cutilSafeCall(hipMemset((void*)device_ww_count.data, 0, sizeof(unsigned int) * device_ww_count.row));
    cutilSafeCall(hipMemset((void*)device_ww_count2.data, 0, sizeof(unsigned int) * device_ww_count2.row));
    cutilSafeCall(hipMemset(device_ww2.data, 0, sizeof(float) * device_ww2.row * device_ww2.col));
    cutilSafeCall(hipMemset(device_ret.data, 0, sizeof(unsigned int) * device_ret.row));


    //this is related to IMAGE_MXN
    calc_ww2<<<IMAGE_MxN/128,128>>>(device_ww,device_ww2.data);
    hipDeviceSynchronize();

    cutilSafeCall(hipMemcpy(device_save.data, device_ww2.data, sizeof(float) * device_ww2.row * device_ww2.col, hipMemcpyDeviceToDevice));
    cublasInit();
    for (int i=0; i<DATA_SIZE; i++){
    	if ( !(i % 10000) )
    		printf("%d\n",i);
	    cutilSafeCall(hipMemcpy(device_ww2.data, device_save.data, sizeof(float) * device_ww2.row * device_ww2.col, hipMemcpyDeviceToDevice));
		hipblasSgemv('T', device_ww.row, device_ww.col, 2, device_ww.data, device_ww.row,
				device_data.data + i * device_data.col,
				1,
				-1,
				device_ww2.data,
				1);
		hipDeviceSynchronize();

		hipError_t lasterror = hipGetLastError();
		if (lasterror)
			printf("sgemv: %s\n", hipGetErrorString(lasterror));

		//the device_ww_count that's returned *might* be transposed.  Right now, the data is correct, but might need tranposing.
    	reduce<<<1,REDUCE_BLOCKSIZE>>>(device_ww_count.data,device_indices.data,device_sum.data, device_ww2.data,device_data.data, i);
    	hipDeviceSynchronize();
    	lasterror = hipGetLastError();
    	if (lasterror)
        	printf("reduce:%d %s\n", i, hipGetErrorString(lasterror));
    }

    cublasShutdown();

    ORDERED_MATRIX<unsigned int, COLUMN_MAJOR> count;
    count.row = 32;
    count.col = 32;
    count.data = (unsigned int*)malloc(count.row * count.col * sizeof(unsigned int));
	hipMemcpy(count.data, device_ww_count.data, device_ww_count.row * device_ww_count.col * sizeof(unsigned int), hipMemcpyDeviceToHost);

	ORDERED_MATRIX<int, COLUMN_MAJOR> cnt;
	cnt.row = 32;
	cnt.col = 32;
	cnt.data = (int*)malloc(cnt.row * cnt.col * sizeof(int));
	memset(cnt.data, 0, sizeof(int) * cnt.row * cnt.col);

	ORDERED_MATRIX<float, COLUMN_MAJOR> argh;
	argh.row = device_sum.row;
	argh.col = device_sum.col;
	argh.data = (float*)malloc(argh.row * argh.col * sizeof(float));
	hipMemcpy(argh.data, device_sum.data, sizeof(float) * argh.col * argh.row, hipMemcpyDeviceToHost);

	ORDERED_MATRIX<float, COLUMN_MAJOR> cc_sum;
	cc_sum.row = 4;
	cc_sum.col = 1024;
	cc_sum.data = (float*)malloc(cc_sum.row * cc_sum.col *sizeof(float));
	memset(cc_sum.data, 0 , sizeof(float) * cc_sum.row * cc_sum.col);
	for (int i=0; i<32; i++){
		int imin = max(i - host_beta[0],0);
		int imax = min(i+ host_beta[0] + 1,IMAGE_N);

		for (int j=0; j<32; j++){
			for (int x=imin; x<imax; x++){
				for (int k=0; k<4; k++){
					cc_sum(k, i + IMAGE_M * j) += argh(k, x + IMAGE_M * j);
				}
				cnt(i,j) += count(x,j);
			}
		}
	}

	memset(argh.data, 0, sizeof(float) * argh.row * argh.col);
	memset(count.data, 0, sizeof(int) * count.row * count.col);

	for (int i=0; i<32; i++){
		int imin = max(i - host_beta[0],0);
		int imax = min(i+ host_beta[0] + 1,IMAGE_N);
		for (int j=0; j<32; j++){
			float sum = 0;
			for (int x=imin; x<imax; x++){
				for (int k=0; k<4; k++){
					argh(k, i + IMAGE_M * j) += cc_sum(k, j + IMAGE_M * x);
				}
				count(j,i) += cnt(j,x);
			}
		}
	}

	hipMemset(device_scratch.data, 0, sizeof(float) * IMAGE_MxN * VECTOR_SIZE);
	grid = dim3(2,2);
	block = dim3(16,16);
	update_weights<<<grid,block>>>(device_sum.data, device_scratch.data, device_ww_count.data, device_ww_count2.data, host_beta[0]);
	hipDeviceSynchronize();

	update_weights2<<<grid,block>>>(device_ww.data, device_sum.data, device_scratch.data, device_ww_count.data, device_ww_count2.data, host_beta[0], host_alpha[0]);
	hipDeviceSynchronize();

	cutilSafeCall(hipMemcpy(cc_sum.data, device_sum.data, cc_sum.row * cc_sum.col * sizeof(float), hipMemcpyDeviceToHost));
		for (int i=0; i<4; i++){
			for (int j=0; j<32; j++){
				for (int k=0; k<32; k++){
					printf("%f ", cc_sum(i, j + IMAGE_M * k));
				}
				printf("\n");
			}
		}

    cutStopTimer(timer);
    time = cutGetTimerValue(timer);
    total_time += time;
    printf("Run time %f\n\n", time);
    cutResetTimer(timer);

    block = dim3(16,16);
    grid = dim3(IMAGE_M/16, IMAGE_N/16);
    buildImage<<<BUILD_IMAGE_GRID_SIZE,32>>>(device_ret.data + GENOMIC_DATA_COUNT * IMAGE_MxN,
    											device_labels.data,device_indices.data);
    for (int i=0; i<GENOMIC_DATA_COUNT; i++)
    	buildSplitImage<<<grid,block>>>(device_ret.data + i * IMAGE_MxN,device_labels.data,device_indices.data,i);

    expandConstantImage<<<grid,block>>>(device_regular_pbo, device_ret.data + GENOMIC_DATA_COUNT * IMAGE_MxN);
	expandLogImage<<<grid,block>>>(device_log_pbo, device_ww_count.data + GENOMIC_DATA_COUNT * IMAGE_MxN);
	generateSplitImage(genome_index, device_split_pbo);


    printf("Total Time: %f\n\n", total_time);
#if DEBUG_PRINT
    unsigned char count[262144];
    cutilSafeCall(hipMemcpy(count, device_log_pbo, sizeof(unsigned char) * 262144, hipMemcpyDeviceToHost));
	int counter = 0;
	for (int i=0; i<512; i++){
		for (int j=0; j<512; j++){
			printf("%d ", count[i * 512 + j]);
		}
		printf("\n");
	}

#endif
	host_r++;
	host_alpha[0] = max(0.01, host_alpha[1] * (1.0 - ((float)host_r/host_T)));
	host_beta[0] = max(0., host_beta[1] - host_r / 1.5);

	printf("r: %d alpha %f: beta %d\n", host_r, host_alpha[0], host_beta[0]);
   	return EXIT_SUCCESS;
}
