#include "shared.h"
#include <hipblas.h>
#include <stdio.h>
#include <cutil_inline.h>



void modify (float *A, int lda, float *B, int ldb, float *C, int ldc, float alpha,float beta, int M, int N)
{
	hipblasSgemm('N','N', M,M,M,
			alpha,
			A, lda,
			B, ldb,
			beta, C, ldc);
	hipblasStatus_t stat = cublasGetError();
	if (stat != HIPBLAS_STATUS_SUCCESS){
		printf("Error # %d: sgemm failed\n", stat);
	}
}

void setupMatrix(float *&device_matrix, float *host_mem, float set_num, int M, int N)
{
    hipblasStatus_t stat;
    stat = cublasAlloc (M*N, sizeof(*host_mem), (void**)&device_matrix);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("device memory allocation failed\n");
    }
    for (int j = 0; j < N; j++) {
        for (int i = 0; i < M; i++) {
        	host_mem[IDX2C(i,j,M)] = set_num;//j * M + i + 1;
        }
    }
    stat = hipblasSetMatrix (M, N, sizeof(*host_mem), host_mem, M, device_matrix, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed\n");
    }

}
extern "C" int runCudasGemm(int M, int N)
{
    hipblasStatus_t stat;
    float* device_A, *device_B, *device_C;
    float* a = 0;
    a = (float *)malloc (M * N * sizeof (*a));
    if (!a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    unsigned int timer;
    cutCreateTimer(&timer);
    double time,total_time;

    total_time = 0;
    cutResetTimer(timer);
    cutStartTimer(timer);
    cublasInit();

    //this isn't any faster...surprising...
//    hipMalloc((void**)&device_A, sizeof(float) * M * N * 3);
//    device_B = device_A + M * N;
//    device_C = device_B + M * N;
    printf("setup matrix A\n");
    setupMatrix(device_A, a, 1, M, N);
    printf("setup matrix B\n");
    setupMatrix(device_B, a, 1, M, N);
    printf("setup matrix C\n");
    setupMatrix(device_C, a, 0, M, N);
    cutStopTimer(timer);
    time = cutGetTimerValue(timer);
    total_time += time;
    printf("Initialization time %f\n\n", time);

    cutResetTimer(timer);
    cutStartTimer(timer);
    modify (device_A, M, device_B, M, device_C, M, 1.0, 0.0, M, N);
    hipDeviceSynchronize();

    cutStopTimer(timer);
    time = cutGetTimerValue(timer);
    total_time += time;

    printf("Run time %f\n\n", time);

    cutResetTimer(timer);
    cutStartTimer(timer);
    stat = hipblasGetMatrix (M, N, sizeof(*a), device_C, M, a, M);

    cutStopTimer(timer);
    time = cutGetTimerValue(timer);
    total_time += time;

    printf("Transfer back time %f\n\n", time);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        cublasFree (device_C);
        cublasShutdown();
        return EXIT_FAILURE;
    }

    printf("Total Time: %f\n\n", total_time);
    cublasFree (device_A);
    cublasFree (device_B);
    cublasFree (device_C);

    cublasShutdown();

    for (int j = 0; j < N; j+=512) {
        for (int i = 0; i < M; i+=512) {
            printf ("%7.0f ", a[IDX2C(i,j,M)]);
        }
        printf ("\n");
    }
    return EXIT_SUCCESS;
  }


