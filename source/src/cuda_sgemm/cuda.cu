#include "hip/hip_runtime.h"
#include "shared.h"
#include <hipblas.h>
#include <stdio.h>
#include <cutil_inline.h>

//there's probably a much nicer way to do this...
//but lets try this for now
__global__ void coalesce(const float *ww, const float *data, float *ww2,
						int *ret,
						float alpha, float beta,
						int M, int N, int K)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	float sum = 0;
	float max_val = -10000;
	int argmax = 0;
	if (i < 20000){
		for (int j=0; j<M; j++){
			sum = 0;
			for (int k=0; k<K; k++){
				sum += ww[j * K + k] * data[k * N + i];
			}
			sum = alpha * sum + beta * ww2[j * N + i];
			//ww2[j * N + i] = sum;
			if (max_val < sum){
					argmax = j;
					max_val = sum;
			}

		}
		ret[argmax]++;
	}
}


//unoptimized this is 10 times slower than calling cublaSgemm
//but I can't figure out what the deal is with sGemm...
//row-major order...
__global__ void sgemm(const float *A, int lda, const float *B, int ldb, float *C, int ldc,
						float alpha, float beta,
						int M, int N, int K)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	float sum = 0;
	for (int k=0; k<K; k++){
		sum += A[i * K + k] * B[k * N + j];

	}

	C[i * N + j] = alpha * sum + beta * C[i * N + j];
}


void setupMatrix(float *&device_matrix, float *host_mem, float set_num, int M, int N)
{
    hipblasStatus_t stat;
    stat = cublasAlloc (M*N, sizeof(*host_mem), (void**)&device_matrix);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("device memory allocation failed\n");
    }
    for (int j = 0; j < N; j++) {
        for (int i = 0; i < M; i++) {
        	host_mem[IDX2C(i,j,M)] = set_num;//j * M + i + 1;
        }
    }
    stat = hipblasSetMatrix (M, N, sizeof(*host_mem), host_mem, M, device_matrix, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed\n");
    }
}

extern "C" int runCudasGemm(MATRIX ww, MATRIX ww2, MATRIX data)
{
    float* device_A, *device_B, *device_C;
    float* a = 0;
    a = (float *)malloc (ww.row * data.col * sizeof (*a));
    if (!a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }

    int *device_ret = 0;
	int *ret = (int*)malloc(sizeof(int) * ww.row);

    unsigned int timer;
    cutCreateTimer(&timer);
    double time,total_time;

    total_time = 0;
    cutResetTimer(timer);
    cutStartTimer(timer);

    printf("setup matrix ww %d %d\n", ww.row, ww.col);
    cutilSafeCall(hipMalloc((void**)&device_A, sizeof(float) * ww.row * ww.col));
    cutilSafeCall(hipMemcpy(device_A, ww.data, sizeof(float) * ww.row * ww.col, hipMemcpyHostToDevice));

    printf("setup matrix data %d %d\n", data.row, data.col);
    cutilSafeCall(hipMalloc((void**)&device_B, sizeof(float) * data.row*data.col));
    cutilSafeCall(hipMemcpy(device_B, data.data, sizeof(float) * data.row * data.col, hipMemcpyHostToDevice));
    //setupMatrix(device_B, a, 1, data.row, data.col);

    printf("setup matrix ww2 %d %d\n", ww.row, data.col);
    cutilSafeCall(hipMalloc((void**)&device_C, sizeof(float) * data.col * ww.row));
    for (int i=0; i<ww.row; i++){
    	for (int j=0; j<data.col; j++){
    		a[i * data.col + j] = ww2.data[i];
    	}
    }
    cutilSafeCall(hipMemcpy(device_C, a, sizeof(float) * ww.row * data.col, hipMemcpyHostToDevice));

    cutilSafeCall(hipMalloc((void**)&device_ret, sizeof(int) * ww.row));
    cutilSafeCall(hipMemset((void*)device_ret, 0, sizeof(int) * ww.row));
    cutStopTimer(timer);
    time = cutGetTimerValue(timer);
    total_time += time;
    printf("Initialization time %f\n\n", time);

    cutResetTimer(timer);
    cutStartTimer(timer);

    coalesce<<<80,256>>>(device_A, device_B, device_C, device_ret, 2.0, -1.0, ww.row,data.col, data.row);
    hipDeviceSynchronize();

    cutStopTimer(timer);
    time = cutGetTimerValue(timer);
    total_time += time;

    printf("Run time %f\n\n", time);

    cutResetTimer(timer);
    cutStartTimer(timer);
    cutilSafeCall(hipMemcpy(ret, device_ret, sizeof(int) * ww.row , hipMemcpyDeviceToHost));

    cutStopTimer(timer);
    time = cutGetTimerValue(timer);
    total_time += time;

    printf("Transfer back time %f\n\n", time);

    printf("Total Time: %f\n\n", total_time);
    hipFree (device_A);
    hipFree (device_B);
    hipFree (device_C);
	hipFree(device_ret);
    for (int i=0; i<5; i++){
    	for (int j=0; j<5; j++){
    		printf("%f ", a[i * data.col +j ]);
    	}
    	printf("\n");
    }

//    int new_ww_count[896];
//	for (int i=0; i< 896; i++){
//		new_ww_count[i] = 0;
//	}
//	int argmax = 0;
//	float max_val = 0;
//	for (int i=0; i<20000; i++){
//			argmax = 0;
//			max_val = -100000;
//			for (int j=0; j<896; j++){
//					if (max_val < a[j * 20000 + i]){
//							argmax = j;
//							max_val = a[j * 20000 + i];
//					}
//			}
//
//			new_ww_count[argmax]++;
//	}

	int counter = 0;
	for (int i=0; i<56; i++){
		 for (int j=0; j<16; j++){
				printf("%d ", ret[i * 16 + j]);
				counter += ret[i * 16 + j];
		 }
		 printf("\n");
	}
	printf("%d\n",counter);


	delete a, ret;
    return EXIT_SUCCESS;
}



