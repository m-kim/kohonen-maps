#include "hip/hip_runtime.h"
#include "shared.h"
#include <hipblas.h>
#include <stdio.h>
#include <cutil_inline.h>
#define EPSILON 0.000001

extern "C" int genome_index;

MATRIX<MATRIX_TYPE> device_ww2, device_save, device_sum, device_scratch;
MATRIX<unsigned int> device_labels, device_indices,device_ww_count, device_ret,device_ww_count2;
ORDERED_MATRIX<MATRIX_TYPE, COLUMN_MAJOR> device_ww, device_data;


unsigned int *ret, *indices;

float host_alpha[2];
int host_r = -1, host_beta[2];

__constant__ uint constant_color[COLOR_SIZE];


__global__ void calc_ww2(const MATRIX<MATRIX_TYPE> ww, MATRIX_TYPE *ww2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int j=0; j<VECTOR_SIZE; j++){
		//this shouldn't be backwards...*sigh*
		ww2[i] += pow(ww.data[i * ww.row + j ],2);
	}
}

__global__ void update_weights(float *a, float *b, uint *ww_count, uint *count, int _beta)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int slab = threadIdx.y + blockDim.y * blockIdx.y;
	int index = j * IMAGE_M + slab;

	if (slab < IMAGE_M){
		int _min = max(j - _beta, 0);
		int _max = min(j + _beta + 1, IMAGE_N);

		for (int i=0; i<VECTOR_SIZE; i++){  //vector size...
			for (int k= _min; k<_max; k++){
				b[i * IMAGE_MxN + index]  += a[i * IMAGE_MxN + k * IMAGE_M + slab];
			}
		}
		for (int k= _min; k<_max; k++){
			count[index] += ww_count[k * IMAGE_M + slab];
		}
	}
}

__global__ void update_weights2(float *ww, float *a, float *b, uint *ww_count, uint *count, int _beta, float _alpha)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int slab = threadIdx.y + blockDim.y * blockIdx.y;
	int index = j * IMAGE_M + slab;
//	__shared__ float s_ww[IMAGE_N * IMAGE_M];
	int _min = max(slab - _beta, 0);
	int _max = min(slab + _beta + 1, IMAGE_M);

	if (slab < IMAGE_M){

		for (int i=0; i<VECTOR_SIZE; i++){  //vector size...
			for (int k= _min; k<_max; k++){
				a[i * IMAGE_MxN + index]  += b[i * IMAGE_MxN + j * IMAGE_M + k];
			}
		}
		for (int k= _min; k<_max; k++){
			ww_count[index] += count[j * IMAGE_M + k];
		}

		for (int i=0; i<VECTOR_SIZE; i++){
			if (ww_count[index] == 0)
				a[i * IMAGE_MxN + index] = 0;
			else
				a[ i * IMAGE_MxN + index] = a[ i * IMAGE_MxN + index] / (ww_count[index] + EPSILON);
        	ww[i * IMAGE_MxN + index] = abs(ww[i * IMAGE_MxN + index]  +_alpha * (a[i * IMAGE_MxN + index] - ww[i * IMAGE_MxN + index]));
		}
		for (int i=0; i<VECTOR_SIZE; i++){
	    	ww[index] += ww[i * IMAGE_MxN + index];
		}
	}
	__syncthreads();
	if (slab < IMAGE_M){
		for (int i=0; i<VECTOR_SIZE; i++){
			if (ww[index] > 0)
				ww[i * IMAGE_MxN + index] = ww[i * IMAGE_MxN + index] / (ww[index]);
			else
				ww[i * IMAGE_MxN + index] = 0;

		}
	}
}

//Calculate argmax and sum the data vectors
__global__ void reduce(uint *ret, uint *indices, float *ww_sum, const float *vec, const float *data, int index)
{
	int size = 1024;
	//using shared memory here will limit me...
	//initialize with hard coded numbers because compile error on variable initialization
	__shared__ int argmax[1024];
	__shared__ float s_vec[1024];

	int blocksize = REDUCE_BLOCKSIZE;
	int coalesce_num = size/blocksize;

	for (int i=0; i<1024/REDUCE_BLOCKSIZE; i++){
		argmax[threadIdx.x + i * blocksize] = threadIdx.x + i * blocksize;
		s_vec[threadIdx.x + i * blocksize] = vec[threadIdx.x + i * blocksize];
	}


	// Large number ->32
	for (int j=1; j < coalesce_num; j++){
		if (threadIdx.x + blocksize * j < IMAGE_MxN){
			argmax[threadIdx.x] = (s_vec[argmax[threadIdx.x]] > s_vec[argmax[j * blocksize + threadIdx.x]])?
						argmax[threadIdx.x]:argmax[j * blocksize + threadIdx.x];
		}
	}

	//32->16, 16->8, 8->4, 4->2, 2->1
	for (int i=0; i<LOG2_REDUCE_BLOCKSIZE; i++){
		__syncthreads();
		blocksize = blocksize/2;

		if (threadIdx.x < blocksize){
			argmax[threadIdx.x] = s_vec[ argmax[blocksize +threadIdx.x]] < s_vec[argmax[threadIdx.x]]? argmax[threadIdx.x]:(argmax[blocksize+threadIdx.x]);
		}
	}
	__syncthreads();
	if (threadIdx.x < 1)
		ret[ argmax[0] ]++;
	indices[index] = argmax[0];

	//take the vector from data and save it to ww_sum
	if (threadIdx.x < VECTOR_SIZE)
		ww_sum[ argmax[0] + threadIdx.x * IMAGE_MxN] += data[index * VECTOR_SIZE + threadIdx.x];//ww_sum[ 410 * 16 + threadIdx.x] = data[threadIdx.x];
}

__global__ void buildImage(uint *im, uint *labels, uint *indices)
{
	uint i = threadIdx.x + blockDim.x * blockIdx.x;
	__shared__ int nn[IMAGE_N];
	__shared__ int mm[IMAGE_N];
	nn[threadIdx.x] = indices[i] / IMAGE_M;
	mm[threadIdx.x] = indices[i] - IMAGE_M * nn[threadIdx.x];
	im[ nn[threadIdx.x] * IMAGE_M + mm[threadIdx.x]] = labels[i] + 1;
}

__global__ void buildSplitImage(uint *im, uint *labels, uint *indices, int g_index)
{
	uint tidx = threadIdx.x + blockDim.x * blockIdx.x;
	uint tidy = threadIdx.y + blockDim.y * blockIdx.y;
	uint index = tidx * IMAGE_N + tidy;

	int genome[GENOMIC_DATA_COUNT];

	for (int i=0; i<GENOMIC_DATA_COUNT; i++)
		genome[i] = 0;

	for (int i=0; i<DATA_SIZE; i++){
		if (indices[i] == index){
			genome[ labels[i] ]++;
		}
	}

	int count = 0;
	for (int i=0; i<GENOMIC_DATA_COUNT; i++){
		count = 0;
		for (int j=0; j<GENOMIC_DATA_COUNT; j++){
			if (i != j)
				count += (genome[i] > genome[j]);
		}
		if (count == (GENOMIC_DATA_COUNT - 1)){
			im[index] = genome[g_index];
			return;
		}
	}
	im[index] = 0;
//	if (genome[0] > genome[1] && genome[0] > genome[2] && genome[0] > genome[3])
//		im[index] = genome[g_index];
//	else if (genome[1] > genome[0] && genome[1] > genome[2] && genome[1] > genome[3])
//		im[index] = genome[g_index];
//	else if (genome[2] > genome[0] && genome[2] > genome[1] && genome[2] > genome[3])
//		im[index] = genome[g_index];
//	else if (genome[3] > genome[0] && genome[3] > genome[1] && genome[3] > genome[2])
//		im[index] = genome[g_index];
//	else
//		im[index] = 0;
}

__global__ void expandSplitImage(uint *im, const uint *ret)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	for (int i=0; i<16; i++){
		for (int j=0; j<16; j++){
			im[(y * 16 + j) * 512 + x * 16 + i] = LUMINANCE_ADJUSTMENT * ret[y * IMAGE_M + x];
		}
	}
}

__global__ void expandLogImage(unsigned char *im, const uint *ret)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	for (int i=0; i<16; i++){
		for (int j=0; j<16; j++){
			im[(y * 16 + j) * 512 + x * 16 + i] = logf(ret[y * IMAGE_M + x]);
		}
	}
}
__global__ void expandConstantImage(uint *im, const uint *ret)
{

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	for (int i=0; i<16; i++){
		for (int j=0; j<16; j++){
			im[(y * 16 + j) * 512 + x * 16 + i] = constant_color[ret[y * IMAGE_M + x]] * ret[y * IMAGE_M + x];
		}
	}
}

extern "C" void generateSplitImage(int g_index, unsigned int * device_split_pbo)
{
	dim3 block(16,16);
	dim3 grid(IMAGE_M/16,IMAGE_N/16);
	expandSplitImage<<<grid,block>>>(device_split_pbo, device_ret.data + g_index * IMAGE_MxN);
}

extern "C" void cleanup()
{
    hipFree (device_ww.data);
    hipFree (device_data.data);
    hipFree (device_ww2.data);
	hipFree(device_indices.data);
	hipFree(device_labels.data);
	delete  ret, indices;
}
extern "C" void setupCuda(ORDERED_MATRIX<MATRIX_TYPE, COLUMN_MAJOR> ww,  ORDERED_MATRIX<MATRIX_TYPE, COLUMN_MAJOR> data, uint *labels, unsigned int *device_regular_pbo, uint *device_split_pbo, unsigned char *device_log_pbo)
{
    //setup color
	unsigned char color[COLOR_SIZE * 4];
	for(unsigned int i=0; i<COLOR_SIZE * 4; i+=4){
		color[i + 1] = (unsigned char)i;
		color[i + 2] = (i + 64) % 256;
		color[i + 3] = (i + 128) % 256;
		color[i] = (i + 192) % 256;
	}

	color[0] = 0;
	color[1] = 0;
	color[2] = 0;
	color[3] = 0;

	color[4] = 255;
	color[5] = 0;
	color[6] = 0;
	color[7] = 0;

	color[8] = 0;
	color[9] = 255;
	color[10] = 0;
	color[11] = 0;

	color[12] = 0;
	color[13] = 0;
	color[14] = 255;
	color[15] = 0;

	color[16] = 255;
	color[17] = 255;
	color[18] = 0;
	color[19] = 0;

	color[20] = 255;
	color[21] = 0;
	color[22] = 255;
	color[23] = 0;

	color[24] = 0;
	color[25] = 255;
	color[26] = 255;
	color[27] = 0;

	color[28] = 128;
	color[29] = 128;
	color[30] = 128;
	color[31] = 0;

	color[32] = 255;
	color[33] = 255;
	color[34] = 255;
	color[35] = 0;

	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(constant_color), color, sizeof(unsigned int) * COLOR_SIZE, 0));

	host_beta[0] = 10;
	host_beta[1] = 10;
	host_alpha[0] = .6;
	host_alpha[1] = .6;


	hipMemset(device_regular_pbo, 128, sizeof(unsigned int) * 512 * 512);
	hipMemset(device_split_pbo, 128, sizeof(unsigned int) * 512 * 512);
	hipMemset(device_log_pbo, 128, sizeof(unsigned char) * 512 * 512);

	device_labels.row = data.row;
	device_labels.col = 1;
	cutilSafeCall(hipMalloc((void**)&device_labels.data, sizeof(uint) * data.row));
	cutilSafeCall(hipMemcpy(device_labels.data, labels, sizeof(uint) * data.row, hipMemcpyHostToDevice));

	device_ww_count.row = ww.row;
	device_ww_count.col = 1;
	cutilSafeCall(hipMalloc((void**)&device_ww_count.data, sizeof(unsigned int) * device_ww_count.row));
    cutilSafeCall(hipMemset((void*)device_ww_count.data, 0, sizeof(unsigned int) * device_ww_count.row));

	device_ww_count2.row = ww.row;
	device_ww_count2.col = ww.col;
	cutilSafeCall(hipMalloc((void**)&device_ww_count2.data, sizeof(unsigned int) * device_ww_count2.row));
    cutilSafeCall(hipMemset((void*)device_ww_count2.data, 0, sizeof(unsigned int) * device_ww_count2.row));

    //multiply by the number of genomes
    //+1 for the regular image
    device_ret.row = ww.row * (GENOMIC_DATA_COUNT + 1);
    device_ret.col = 1;
	cutilSafeCall(hipMalloc((void**)&device_ret.data, sizeof(unsigned int) * device_ret.row));
    cutilSafeCall(hipMemset((void*)device_ret.data, 0, sizeof(unsigned int) * device_ret.row));

    device_indices.row = DATA_SIZE;
    device_indices.col = 1;
    cutilSafeCall(hipMalloc((void**)&device_indices.data, sizeof(unsigned int) * DATA_SIZE));
    cutilSafeCall(hipMemset((void*)device_indices.data, 0, sizeof(unsigned int) * DATA_SIZE));

    device_ww.row = ww.row;
    device_ww.col = ww.col;
    printf("setup matrix ww %d %d\n", ww.row, ww.col);
    cutilSafeCall(hipMalloc((void**)&device_ww.data, sizeof(float) * ww.row * ww.col));
    cutilSafeCall(hipMemcpy(device_ww.data, ww.data, sizeof(float) * ww.row * ww.col, hipMemcpyHostToDevice));

	device_ww2.row = IMAGE_N;
	device_ww2.col = IMAGE_M;
    printf("setup matrix ww2 %d %d\n", device_ww2.row, device_ww2.col);
    cutilSafeCall(hipMalloc((void**)&device_ww2.data, sizeof(float) * device_ww2.row * device_ww2.col));
	cutilSafeCall(hipMemset(device_ww2.data, 0, sizeof(float) * device_ww2.row * device_ww2.col));

    device_sum.row = ww.row;
    device_sum.col = ww.col;
    printf("setup matrix sum %d %d\n", device_sum.row, device_sum.col);
    cutilSafeCall(hipMalloc((void**)&device_sum.data, sizeof(float) * device_sum.row * device_sum.col));
    cutilSafeCall(hipMemset(device_sum.data, 0, sizeof(float) * device_sum.row * device_sum.col ));


    printf("setup matrix scractch %d %d\n", IMAGE_MxN, VECTOR_SIZE);
    cutilSafeCall(hipMalloc((void**)&device_scratch.data, sizeof(float) * IMAGE_MxN * VECTOR_SIZE));
    cutilSafeCall(hipMemset(device_scratch.data, 0, sizeof(float) * IMAGE_MxN * VECTOR_SIZE));

    device_save.row = device_ww2.row;
    device_save.col = device_ww2.col;
    device_save.data = device_scratch.data;


	ret = (unsigned int*)malloc(sizeof(unsigned int) * ww.row);
	indices = (uint*)malloc(sizeof(uint) * data.row);

    device_data.row = data.row;
    device_data.col = data.col;
    printf("setup matrix data %d %d\n", device_data.row, device_data.col);
    cutilSafeCall(hipMalloc((void**)&device_data.data, sizeof(float) * device_data.row*device_data.col));
    cutilSafeCall(hipMemcpy(device_data.data, data.data, sizeof(float) * device_data.row * device_data.col, hipMemcpyHostToDevice));
//    for (int i=0; i<data.row; i++){
//    	for (int j=0; j<data.col; j++){
//    		cutilSafeCall(hipMemcpy(
//    				device_data.data  + (j * data.row + i),
//    				data.data + (i * data.col + j), sizeof(float), hipMemcpyHostToDevice));
//    	}
//    }
}

extern "C" int runCuda(unsigned int *device_regular_pbo, unsigned int *device_split_pbo, unsigned char *device_log_pbo)
{
	unsigned int timer;
    cutCreateTimer(&timer);
    double time,total_time;

    dim3 block;
    dim3 grid;

    total_time = 0;
    cutResetTimer(timer);
    cutStartTimer(timer);
    cutilSafeCall(hipMemset((void*)device_ww_count.data, 0, sizeof(unsigned int) * device_ww_count.row));
    cutilSafeCall(hipMemset((void*)device_ww_count2.data, 0, sizeof(unsigned int) * device_ww_count2.row));

    hipMemset(device_ww2.data, 0, sizeof(float) * device_ww2.row * device_ww2.col);

    hipMemset(device_ret.data, 0, sizeof(unsigned int) * device_ret.row);
    //this is related to IMAGE_MXN
    calc_ww2<<<IMAGE_MxN/128,128>>>(device_ww,device_ww2.data);
    hipDeviceSynchronize();

    ORDERED_MATRIX<float, COLUMN_MAJOR> tmp;
    tmp.row = 1024;
    tmp.col = 1;

    tmp.data = (float*)malloc(sizeof(float) * 1024);

    cutilSafeCall(hipMemcpy(device_save.data, device_ww2.data, sizeof(float) * device_ww2.row * device_ww2.col, hipMemcpyDeviceToDevice));
    cublasInit();
    for (int i=0; i<1; i++){
    	if ( !(i % 10000) )
    		printf("%d\n",i);
	    cutilSafeCall(hipMemcpy(device_ww2.data, device_save.data, sizeof(float) * device_ww2.row * device_ww2.col, hipMemcpyDeviceToDevice));
		hipblasSgemv('T', device_ww.row, device_ww.col, 1, device_ww.data, device_ww.row,
				device_data.data + i * device_data.col,
				1,
				0,
				device_ww2.data,
				1);
		hipDeviceSynchronize();
		hipMemcpy(tmp.data, device_ww2.data, sizeof(float) * 1024, hipMemcpyDeviceToHost);
		tmp.print();
		hipError_t lasterror = hipGetLastError();
		if (lasterror)
			printf("sgemv: %s\n", hipGetErrorString(lasterror));
    	reduce<<<1,REDUCE_BLOCKSIZE>>>(device_ww_count.data,device_indices.data,device_sum.data, device_ww2.data,device_data.data, i);
    	hipDeviceSynchronize();
    	lasterror = hipGetLastError();
    	if (lasterror)
        	printf("reduce:%d %s\n", i, hipGetErrorString(lasterror));
    }

    cublasShutdown();

    block = dim3(16,16);
    grid = dim3(2,2);
    buildImage<<<BUILD_IMAGE_GRID_SIZE,32>>>(device_ret.data + GENOMIC_DATA_COUNT * IMAGE_MxN,
    											device_labels.data,device_indices.data);
	hipDeviceSynchronize();
	block = dim3(16,16);
	grid = dim3(IMAGE_M/16,IMAGE_N/16);
	expandConstantImage<<<grid,block>>>(device_regular_pbo, device_ret.data + GENOMIC_DATA_COUNT * IMAGE_MxN);

    printf("build image %s\n", hipGetErrorString(hipGetLastError()));

    for (int i=0; i<GENOMIC_DATA_COUNT; i++)
    	buildSplitImage<<<grid,block>>>(device_ret.data + i * IMAGE_MxN,device_labels.data,device_indices.data,i);


    hipDeviceSynchronize();
    printf("build split image %s\n", hipGetErrorString(hipGetLastError()));
    cutStopTimer(timer);
    time = cutGetTimerValue(timer);
    total_time += time;
    printf("Run time %f\n\n", time);

    cutResetTimer(timer);

    block = dim3(16,16);
    grid = dim3(IMAGE_M/16, IMAGE_N/16);
    hipMemset(device_scratch.data, 0, sizeof(float) * IMAGE_MxN * VECTOR_SIZE);
	update_weights<<<grid,block>>>(device_sum.data, device_scratch.data, device_ww_count.data, device_ww_count2.data, host_beta[0]);
	hipDeviceSynchronize();
	update_weights2<<<grid,block>>>(device_ww.data, device_sum.data, device_scratch.data, device_ww_count.data, device_ww_count2.data, host_beta[0], host_alpha[0]);

	expandLogImage<<<grid,block>>>(device_log_pbo, device_ww_count.data + GENOMIC_DATA_COUNT * IMAGE_MxN);

	hipDeviceSynchronize();

	generateSplitImage(genome_index, device_split_pbo);


    printf("Total Time: %f\n\n", total_time);
#if DEBUG_PRINT
    unsigned char count[262144];
    cutilSafeCall(hipMemcpy(count, device_log_pbo, sizeof(unsigned char) * 262144, hipMemcpyDeviceToHost));
	int counter = 0;
	for (int i=0; i<512; i++){
		for (int j=0; j<512; j++){
			printf("%d ", count[i * 512 + j]);
		}
		printf("\n");
	}

#endif
	host_r++;
	host_alpha[0] = max(0.01, host_alpha[1] * (1.0 - ((float)host_r/host_T)));
	host_beta[0] = max(0., host_beta[1] - host_r / 1.5);

	printf("r: %d alpha %f: beta %d\n", host_r, host_alpha[0], host_beta[0]);
   	return EXIT_SUCCESS;
}
